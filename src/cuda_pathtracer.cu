#include "hip/hip_runtime.h"
#include "cuda_pathtracer.hpp"
#include "cuda.hpp"
#include "cuda_ray.hpp"
#include "cuda_scene.hpp"
#include "cuda_image.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void kernel();
__device__ void render();
__device__ Color trace();


__global__ void kernel()
{
  //d_render(*image);
}

__device__ void render()
{
  /*
	Vector3 left, right, bottom, top;
	const float aspect = (float) image.resolution.h / image.resolution.w;
	d_camera->calculateFrustumVectors(aspect, &left, &right, &bottom, &top);

	float max_x = (float) image.resolution.w;
	float max_y = (float) image.resolution_org.h;

	long local_y = ray_t::dim::h * blockIdx.y;
	long x = ray_t::dim::w * blockIdx.x;

	auto y = image.getGlobalY(local_y);
  
	auto direction = Vector3((top * (1.f - (2 * y + 1) / max_y) + left * (1.f - (2 * x + 1) / max_x) + d_camera->direction).normalize());
	ray_t ray(d_camera->position, direction);
	
	Color c = trace(ray);
  
	image.setPixel(x, local_y, c);
  */
}

__device__ Color trace() // ray_t ray, unsigned depth
{
  /*
  typename ray_t::distance_t intersection_distance;
  const auto intersected_triangle = d_bih_intersect(ray, &intersection_distance);

  if (d_bih_isNoIntersection(intersected_triangle))
    return scene.background_color;

  const auto &triangle = d_triangles[intersected_triangle];
  const auto material_index = triangle.material_index;
  auto &material = d_materials[material_index];

  if (material.isEmission)
    return material.color;
  if (depth >= opts.max_depth)
    return Color {0, 0, 0};

  const auto N = ray_t::getNormals(scene, intersected_triangle);
  const auto X = (triangle.vertices[1] - triangle.vertices[0]).normalize();
  const auto Y = N.cross(X).normalize();
  const auto &Z = N;
  const auto P = ray.getIntersectionPoint(intersection_distance) + N * 0.0001f;

  Color value{0, 0, 0};
  for (unsigned i = 0; i < opts.num_samples; ++i) {
    ray_t next(P, sampleHemisphere(X, Y, Z));
    value += trace(scene, next, depth + 1);
  }
  value = material.color * value / opts.num_samples;
  return value;
  */
  
  return Color();
}

template<class ray_t, class accel_t>
void CudaPathTracer<ray_t, accel_t>::render(ImageView &image)
{
	CudaImage cudaImage(image);
	cudaImage.initialize();
	
	// copy objects to device
	CudaImage* d_image = cuda::create<CudaImage>(cudaImage);
	
	// configure execution
	// max. 1024 Threads per Block (may ask API)
	// Blocks are assigned to GPU processors -> if there are less GPUs than blocks, one GPU has to calculate several blocks
	dim3 dimGrid(image.resolution.w, image.resolution.h); // BLOCKS per grid (size of a grid)
	
	// start kernel
	//      <<< BLOCKS, THREADS >>>
	kernel<<<dimGrid, 1>>>();
  cuda::checkForError(__FILE__, __func__, __LINE__);
	
	// destroy objects on device
	cuda::destroy<CudaImage>(d_image);
	
	// copy back and free image memory
	cudaImage.finalize();
}

template<class ray_t, class accel_t>
void CudaPathTracer<ray_t, accel_t>::initialize()
{
	accel_cuda_t cudaAccel(accel);
	cudaAccel.initialize();
	
	CudaScene<material_t> cudaScene(scene);
	cudaScene.initialize();
}

template<class ray_t, class accel_t>
void CudaPathTracer<ray_t, accel_t>::finalize()
{
	accel_cuda_t cudaAccel(accel);
	cudaAccel.finalize();
	
	CudaScene<material_t> cudaScene(scene);
	cudaScene.finalize();
}

template class CudaPathTracer<Ray<Scene<EmissionMaterial> >, Bih<Ray<Scene<EmissionMaterial> >, Scene<EmissionMaterial> > >;
